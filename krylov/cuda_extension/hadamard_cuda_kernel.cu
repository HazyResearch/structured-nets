#include "hip/hip_runtime.h"
/* Adated from the CUDA samples https://docs.nvidia.com/cuda/cuda-samples/index.html.
   Changed from "natural order" Hadamard transform (larger strides before
   smaller strides) to the standard Hadamard transform (smaller strides before
   larger strides).
 */

/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;


///////////////////////////////////////////////////////////////////////////////
// Elementary(for vectors less than elementary size) in-shared memory
// combined radix-2 + radix-4 Fast Walsh Transform
///////////////////////////////////////////////////////////////////////////////
#define ELEMENTARY_LOG2SIZE 11

__global__ void fwtBatch1Kernel(float *d_Output, float *d_Input, int log2N)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    const int    N = 1 << log2N;
    const int base = blockIdx.x << log2N;

    //(2 ** 11) * 4 bytes == 8KB -- maximum s_data[] size for G80
    extern __shared__ float s_data[];
    float *d_Src = d_Input  + base;
    float *d_Dst = d_Output + base;

    for (int pos = threadIdx.x; pos < N; pos += blockDim.x)
    {
        s_data[pos] = d_Src[pos];
    }

    int stride = 1;
    //Do single radix-2 stage for odd power of two
    if (log2N & 1)
    {
        cg::sync(cta);

        for (int pos = threadIdx.x; pos < N / 2; pos += blockDim.x)
        {
            int i0 = pos << 1;
            int i1 = i0 + 1;

            float D0 = s_data[i0];
            float D1 = s_data[i1];
            s_data[i0] = D0 + D1;
            s_data[i1] = D0 - D1;
        }
        stride <<= 1;
    }

    //Main radix-4 stages
    const int pos = threadIdx.x;

    for (; stride <= N >> 2; stride <<= 2)
    {
        int lo = pos & (stride - 1);
        int i0 = ((pos - lo) << 2) + lo;
        int i1 = i0 + stride;
        int i2 = i1 + stride;
        int i3 = i2 + stride;

        cg::sync(cta);
        float D0 = s_data[i0];
        float D1 = s_data[i1];
        float D2 = s_data[i2];
        float D3 = s_data[i3];

        float T;
        T = D0;
        D0         = D0 + D2;
        D2         = T - D2;
        T = D1;
        D1         = D1 + D3;
        D3         = T - D3;
        T = D0;
        s_data[i0] = D0 + D1;
        s_data[i1] = T - D1;
        T = D2;
        s_data[i2] = D2 + D3;
        s_data[i3] = T - D3;
    }

    cg::sync(cta);

    for (int pos = threadIdx.x; pos < N; pos += blockDim.x)
    {
        d_Dst[pos] = s_data[pos];
    }
}

////////////////////////////////////////////////////////////////////////////////
// Single in-global memory radix-4 Fast Walsh Transform pass
// (for strides exceeding elementary vector size)
////////////////////////////////////////////////////////////////////////////////
__global__ void fwtBatch2Kernel(
    float *d_Output,
    float *d_Input,
    int stride
)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int   N = blockDim.x *  gridDim.x * 4;

    float *d_Src = d_Input  + blockIdx.y * N;
    float *d_Dst = d_Output + blockIdx.y * N;

    int lo = pos & (stride - 1);
    int i0 = ((pos - lo) << 2) + lo;
    int i1 = i0 + stride;
    int i2 = i1 + stride;
    int i3 = i2 + stride;

    float D0 = d_Src[i0];
    float D1 = d_Src[i1];
    float D2 = d_Src[i2];
    float D3 = d_Src[i3];

    float T;
    T = D0;
    D0        = D0 + D2;
    D2        = T - D2;
    T = D1;
    D1        = D1 + D3;
    D3        = T - D3;
    T = D0;
    d_Dst[i0] = D0 + D1;
    d_Dst[i1] = T - D1;
    T = D2;
    d_Dst[i2] = D2 + D3;
    d_Dst[i3] = T - D3;
}

////////////////////////////////////////////////////////////////////////////////
// Put everything together: batched Fast Walsh Transform CPU front-end
////////////////////////////////////////////////////////////////////////////////
void fwtBatchGPU(float *d_Data, int batchSize, int log2N)
{
    int nMixedRadixPasses = log2N > ELEMENTARY_LOG2SIZE ? ELEMENTARY_LOG2SIZE - (log2N - ELEMENTARY_LOG2SIZE) % 2 : log2N;
    int N = 1 << nMixedRadixPasses;
    int curBatchSize = batchSize << (log2N - nMixedRadixPasses);

    fwtBatch1Kernel<<<curBatchSize, N / 4, N * sizeof(float)>>>(
        d_Data,
        d_Data,
        nMixedRadixPasses
    );

    const int THREAD_N = 256;
    dim3 grid((1 << log2N) / (4 * THREAD_N), batchSize, 1);

    for (int logSize = nMixedRadixPasses + 2; logSize <= log2N; logSize += 2)
    {
        fwtBatch2Kernel<<<grid, THREAD_N>>>(d_Data, d_Data, (1 << logSize) / 4);
    }

}
